
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>  

__global__ void sum_cuda(double* a, double *s, int width) {
  int t = threadIdx.x;
  int b = blockIdx.x*blockDim.x;
  double o[1024];  

  if(b+t < width)
    o[t] = a[b+t];

  __syncthreads();
  
  int i;
  for(i = blockDim.x/2; i > 0; i /= 2) {
    if(t < i && b+t+i < width)
      o[t] += o[t+i];
    
    __syncthreads();
  }

  if(t == 0)
    s[blockIdx.x] = o[0];  
} 

int main()
{
  int width = 40000000;
  int size = width * sizeof(double);

  int block_size = 1024;
  int num_blocks = (width-1)/block_size+1;
  int s_size = (num_blocks * sizeof(double));  
 
  double *a = (double*) malloc (size);
  double *s = (double*) malloc (s_size);

  for(int i = 0; i < width; i++)
    a[i] = i;

  double *d_a, *d_s;

  // alocação e cópia dos dados
  hipMalloc((void **) &d_a, size);
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

  hipMalloc((void **) &d_s, s_size);

  // definição do número de blocos e threads
  dim3 dimGrid(num_blocks,1,1);
  dim3 dimBlock(block_size,1,1);

  // para armazenar o tempo de execução do código
  double time_spent_sum = 0.0;
  double time_spent_copy = 0.0;

  clock_t begin_sum = clock();
  // chamada do kernel
  sum_cuda<<<dimGrid,dimBlock>>>(d_a, d_s, width);
  clock_t end_sum = clock();
  
  // cópia dos resultados para o host
  clock_t begin_copy = clock();
  hipMemcpy(s, d_s, s_size, hipMemcpyDeviceToHost);
  clock_t end_copy = clock();
  // soma das reduções parciais
  for(int i = 1; i < num_blocks; i++) 
    s[0] += s[i];

  printf("\nSum = %f\n",s[0]);
  
  time_spent_sum += (double)(end_sum - begin_sum) / CLOCKS_PER_SEC;
  time_spent_copy += (double)(end_copy - begin_copy) / CLOCKS_PER_SEC;
  printf("\n\nThe elapsed on sum time is %f seconds", time_spent_sum);
  printf("\n\nThe elapsed on copy time is %f seconds\n", time_spent_copy);
  hipFree(d_a);
  hipFree(d_s);
}
