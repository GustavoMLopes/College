
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
/*
 - Tempos
    - GPU:7.412 segundos
    - CPU Sequencial: 2 min 1.036 segundos
    - CPU Paralela: 56 segundos
*/
/*Código PARA GPU - CUDA*/
__global__ void mm(double *a, double *b, double *c, int width)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < width && j < width) {
        double sum = 0;
        for (int k = 0; k < width; k++)
        {
            double x = a[i * width + k];
            double y = b[k * width + j];
            sum += x * y;
        }
        c[i * width + j] = sum;
    }
}

int main()
{
    int width = 2000;
    double *a, *b, *c;
    double *d_a, *d_b, *d_c;

    int size = width * width * sizeof(double);

    a = (double *)malloc(size);
    b = (double *)malloc(size);
    c = (double *)malloc(size);

    for (int i = 0; i < width; i++)
    {
        for (int j = 0; j < width; j++)
        {
            a[i * width + j] = i;
            b[i * width + j] = j;
            c[i * width + j] = 0;
        }
    }

    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(32, 32);
    dim3 numBlocks((width + threadsPerBlock.x - 1) / threadsPerBlock.x, (width + threadsPerBlock.y - 1) / threadsPerBlock.y);

    mm<<<numBlocks, threadsPerBlock>>>(d_a, d_b, d_c, width);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    for(int i = 0; i < width; i++) {
        for(int j = 0; j < width; j++) {
            printf("\n c[%d][%d] = %f",i,j,c[i*width+j]);
        }
    }

    //desalocação de memória
    free(a);
    free(b);
    free(c);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
